#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>
#include <opencv2/xphoto.hpp>
#include <iostream>

#define BLOCK_SIZE 16
#define MAX_INTENSITY_BINS 256

using namespace cv;
using namespace cv::xphoto;

// CUDA Kernel: Color Quantization (Reduce Colors)
__global__ void colorQuantizationKernel(unsigned char *input, unsigned char *output, int width, int height, int levels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int idx = (y * width + x) * 3;
        for (int i = 0; i < 3; i++) {
            output[idx + i] = (input[idx + i] / levels) * levels;
        }
    }
}

// CUDA Kernel: Grayscale Conversion
__global__ void grayscaleKernel(unsigned char *input, unsigned char *output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int idx = (y * width + x) * 3;
        unsigned char r = input[idx];
        unsigned char g = input[idx + 1];
        unsigned char b = input[idx + 2];
        unsigned char gray = (r * 0.299 + g * 0.587 + b * 0.114);
        output[y * width + x] = gray;
    }
}

// CUDA Kernel: Inversion
__global__ void invertKernel(unsigned char *input, unsigned char *output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int idx = y * width + x;
        output[idx] = 255 - input[idx];
    }
}

// CUDA Kernel: Gaussian Blur (Simplified, not optimized)
__global__ void gaussianBlurKernel(unsigned char *input, unsigned char *output, int width, int height, int radius) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int idx = y * width + x;
        float sum = 0.0f;
        int count = 0;
        for (int dy = -radius; dy <= radius; dy++) {
            for (int dx = -radius; dx <= radius; dx++) {
                int nx = x + dx;
                int ny = y + dy;
                if (nx >= 0 && nx < width && ny >= 0 && ny < height) {
                    sum += input[ny * width + nx];
                    count++;
                }
            }
        }
        output[idx] = (unsigned char)(sum / count);
    }
}

// CUDA Kernel: Color Dodge Blending
__global__ void colorDodgeKernel(unsigned char *gray, unsigned char *blur, unsigned char *output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int idx = y * width + x;
        float result = gray[idx] / (1.0f - (blur[idx] / 255.0f));
        output[idx] = (unsigned char)min(result, 255.0f);
    }
}

// CUDA Kernel: Oil Painting Effect
__global__ void oilPaintingKernel(unsigned char *input, unsigned char *output, int width, int height, int radius, int intensityLevels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int idx = (y * width + x) * 3;
        
        // Initialize intensity bins
        int intensityCount[MAX_INTENSITY_BINS] = {0};
        int averageR[MAX_INTENSITY_BINS] = {0};
        int averageG[MAX_INTENSITY_BINS] = {0};
        int averageB[MAX_INTENSITY_BINS] = {0};
        
        // Calculate intensity for neighboring pixels
        for (int dy = -radius; dy <= radius; dy++) {
            for (int dx = -radius; dx <= radius; dx++) {
                int nx = x + dx;
                int ny = y + dy;
                if (nx >= 0 && nx < width && ny >= 0 && ny < height) {
                    int nIdx = (ny * width + nx) * 3;
                    unsigned char r = input[nIdx];
                    unsigned char g = input[nIdx + 1];
                    unsigned char b = input[nIdx + 2];
                    int curIntensity = (int)((double)((r + g + b) / 3) * intensityLevels) / 255;
                    if (curIntensity >= 0 && curIntensity < MAX_INTENSITY_BINS) {
                        intensityCount[curIntensity]++;
                        averageR[curIntensity] += r;
                        averageG[curIntensity] += g;
                        averageB[curIntensity] += b;
                    }
                }
            }
        }
        
        // Find the most populated intensity bin
        int maxCount = 0;
        int maxIndex = 0;
        for (int i = 0; i < MAX_INTENSITY_BINS; i++) {
            if (intensityCount[i] > maxCount) {
                maxCount = intensityCount[i];
                maxIndex = i;
            }
        }
        
        // Calculate final color
        if (maxCount > 0) {
            unsigned char finalR = (unsigned char)(averageR[maxIndex] / maxCount);
            unsigned char finalG = (unsigned char)(averageG[maxIndex] / maxCount);
            unsigned char finalB = (unsigned char)(averageB[maxIndex] / maxCount);
            output[idx] = finalR;
            output[idx + 1] = finalG;
            output[idx + 2] = finalB;
        } else {
            // Handle case where no neighbors are found
            output[idx] = input[idx];
            output[idx + 1] = input[idx + 1];
            output[idx + 2] = input[idx + 2];
        }
    }
}

// CUDA Kernel: Sepia Effect
__global__ void sepiaKernel(unsigned char *input, unsigned char *output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int idx = (y * width + x) * 3;
        unsigned char r = input[idx];
        unsigned char g = input[idx + 1];
        unsigned char b = input[idx + 2];
        unsigned char newR = (unsigned char)(0.393 * r + 0.769 * g + 0.189 * b);
        unsigned char newG = (unsigned char)(0.349 * r + 0.686 * g + 0.168 * b);
        unsigned char newB = (unsigned char)(0.272 * r + 0.534 * g + 0.131 * b);
        output[idx] = newR;
        output[idx + 1] = newG;
        output[idx + 2] = newB;
    }
}

// CUDA Kernel: Negative Effect
__global__ void negativeKernel(unsigned char *input, unsigned char *output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int idx = (y * width + x) * 3;
        output[idx] = 255 - input[idx];
        output[idx + 1] = 255 - input[idx + 1];
        output[idx + 2] = 255 - input[idx + 2];
    }
}

void applyCartoonEffect(Mat &image) {
    Mat smoothImage;
    bilateralFilter(image, smoothImage, 9, 75, 75);
    
    Mat grayImage;
    cvtColor(smoothImage, grayImage, COLOR_BGR2GRAY);
    
    Mat edges;
    Sobel(grayImage, edges, CV_8U, 1, 1);
    threshold(edges, edges, 50, 255, THRESH_BINARY_INV);
    
    int imgSize = image.rows * image.cols * 3;
    unsigned char *d_input, *d_output;
    hipMalloc(&d_input, imgSize);
    hipMalloc(&d_output, imgSize);
    hipMemcpy(d_input, smoothImage.data, imgSize, hipMemcpyHostToDevice);
    
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((image.cols + BLOCK_SIZE - 1) / BLOCK_SIZE, (image.rows + BLOCK_SIZE - 1) / BLOCK_SIZE);
    colorQuantizationKernel<<<grid, block>>>(d_input, d_output, image.cols, image.rows, 64);
    
    Mat quantizedImage(image.rows, image.cols, CV_8UC3);
    hipMemcpy(quantizedImage.data, d_output, imgSize, hipMemcpyDeviceToHost);
    
    Mat cartoonImage;
    bitwise_and(quantizedImage, quantizedImage, cartoonImage, edges);
    
    hipFree(d_input);
    hipFree(d_output);
    
    image = cartoonImage; // Update the original image with the cartoon effect
}

void applyOilPaintingEffect(Mat &image) {
    int imgSize = image.rows * image.cols * 3;
    unsigned char *d_input, *d_output;
    hipMalloc(&d_input, imgSize);
    hipMalloc(&d_output, imgSize);
    hipMemcpy(d_input, image.data, imgSize, hipMemcpyHostToDevice);
    
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((image.cols + BLOCK_SIZE - 1) / BLOCK_SIZE, (image.rows + BLOCK_SIZE - 1) / BLOCK_SIZE);
    oilPaintingKernel<<<grid, block>>>(d_input, d_output, image.cols, image.rows, 5, 20);
    
    hipMemcpy(image.data, d_output, imgSize, hipMemcpyDeviceToHost);
    
    hipFree(d_input);
    hipFree(d_output);
}

void applyManualPencilSketchEffect(Mat &image) {
    int imgSize = image.rows * image.cols * 3;
    unsigned char *d_input;
    hipMalloc(&d_input, imgSize);
    hipMemcpy(d_input, image.data, imgSize, hipMemcpyHostToDevice);
    
    // Grayscale Conversion
    unsigned char *d_gray;
    hipMalloc(&d_gray, image.rows * image.cols);
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((image.cols + BLOCK_SIZE - 1) / BLOCK_SIZE, (image.rows + BLOCK_SIZE - 1) / BLOCK_SIZE);
    grayscaleKernel<<<grid, block>>>(d_input, d_gray, image.cols, image.rows);
    
    // Inversion
    unsigned char *d_inv;
    hipMalloc(&d_inv, image.rows * image.cols);
    invertKernel<<<grid, block>>>(d_gray, d_inv, image.cols, image.rows);
    
    // Gaussian Blur
    unsigned char *d_blur;
    hipMalloc(&d_blur, image.rows * image.cols);
    gaussianBlurKernel<<<grid, block>>>(d_inv, d_blur, image.cols, image.rows, 10);
    
    // Color Dodge Blending
    unsigned char *d_sketch;
    hipMalloc(&d_sketch, image.rows * image.cols);
    colorDodgeKernel<<<grid, block>>>(d_gray, d_blur, d_sketch, image.cols, image.rows);
    
    // Create a new grayscale Mat to hold the sketch
    Mat sketch(image.rows, image.cols, CV_8UC1);
    hipMemcpy(sketch.data, d_sketch, image.rows * image.cols, hipMemcpyDeviceToHost);
    
    // Convert the grayscale sketch to BGR
    cvtColor(sketch, image, COLOR_GRAY2BGR); // Update the original image
    
    hipFree(d_input);
    hipFree(d_gray);
    hipFree(d_inv);
    hipFree(d_blur);
    hipFree(d_sketch);
}

void applySepiaEffect(Mat &image) {
    int imgSize = image.rows * image.cols * 3;
    unsigned char *d_input, *d_output;
    hipMalloc(&d_input, imgSize);
    hipMalloc(&d_output, imgSize);
    hipMemcpy(d_input, image.data, imgSize, hipMemcpyHostToDevice);
    
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((image.cols + BLOCK_SIZE - 1) / BLOCK_SIZE, (image.rows + BLOCK_SIZE - 1) / BLOCK_SIZE);
    sepiaKernel<<<grid, block>>>(d_input, d_output, image.cols, image.rows);
    
    hipMemcpy(image.data, d_output, imgSize, hipMemcpyDeviceToHost);
    
    hipFree(d_input);
    hipFree(d_output);
}

void applyNegativeEffect(Mat &image) {
    int imgSize = image.rows * image.cols * 3;
    unsigned char *d_input, *d_output;
    hipMalloc(&d_input, imgSize);
    hipMalloc(&d_output, imgSize);
    hipMemcpy(d_input, image.data, imgSize, hipMemcpyHostToDevice);
    
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((image.cols + BLOCK_SIZE - 1) / BLOCK_SIZE, (image.rows + BLOCK_SIZE - 1) / BLOCK_SIZE);
    negativeKernel<<<grid, block>>>(d_input, d_output, image.cols, image.rows);
    
    hipMemcpy(image.data, d_output, imgSize, hipMemcpyDeviceToHost);
    
    hipFree(d_input);
    hipFree(d_output);
}

int main() {
    VideoCapture cap(0); // Open default webcam
    if (!cap.isOpened()) {
        std::cerr << "Error opening video stream or file" << std::endl;
        return -1;
    }

    int choice = 0; // Default: No effect

    while (true) {
        Mat frame;
        cap >> frame; // Capture frame

        if (frame.empty()) {
            std::cerr << "Failed to capture frame." << std::endl;
            continue; // Skip processing this iteration
        }


        switch (choice) {
            case 0: { 
                imshow("Webcam", frame); // Display original feed
                break;
            }
            case 1: { 
                Mat tempFrame = frame.clone();
                applyCartoonEffect(tempFrame);
                imshow("Webcam", tempFrame);
                break;
            }
            case 2: { 
                applyOilPaintingEffect(frame);
                imshow("Webcam", frame);
                break;
            }
            case 3: { 
                applyManualPencilSketchEffect(frame);
                imshow("Webcam", frame);
                break;
            }
            case 4: { 
                applySepiaEffect(frame);
                imshow("Webcam", frame);
                break;
            }
            case 5: { 
                applyNegativeEffect(frame);
                imshow("Webcam", frame);
                break;
            }
            default:
                imshow("Webcam", frame); // Display original feed
        }

        char c = (char)waitKey(10);
        if (c == '0') choice = 0; // Default feed
        else if (c == '1') choice = 1; // Cartoon effect
        else if (c == '2') choice = 2; // Oil painting effect
        else if (c == '3') choice = 3; // Pencil sketch effect
        else if (c == '4') choice = 4; // Sepia effect
        else if (c == '5') choice = 5; // Negative effect
        else if (c == 27) break; // ESC to exit
    }

    cap.release();
    destroyAllWindows();
    return 0;
}
