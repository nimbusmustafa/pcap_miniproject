#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>
#include <iostream>
#include <chrono>

#define BLOCK_SIZE 16
#define RADIUS 5  // Neighborhood size for oil painting effect
#define INTENSITY_LEVELS 256

using namespace cv;
using namespace std;
using namespace std::chrono;

// CUDA Kernel: Oil Painting Effect
__global__ void oilPaintingKernel(unsigned char *input, unsigned char *output, int width, int height, int channels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    int hist[INTENSITY_LEVELS] = {0};
    int avgColor[INTENSITY_LEVELS][3] = {0};

    for (int dy = -RADIUS; dy <= RADIUS; dy++) {
        for (int dx = -RADIUS; dx <= RADIUS; dx++) {
            int nx = min(max(x + dx, 0), width - 1);
            int ny = min(max(y + dy, 0), height - 1);
            int index = (ny * width + nx) * channels;

            int intensity = (input[index] + input[index + 1] + input[index + 2]) / 3;
            hist[intensity]++;
            for (int c = 0; c < 3; c++) {
                avgColor[intensity][c] += input[index + c];
            }
        }
    }

    int maxIntensity = 0, maxCount = 0;
    for (int i = 0; i < INTENSITY_LEVELS; i++) {
        if (hist[i] > maxCount) {
            maxCount = hist[i];
            maxIntensity = i;
        }
    }

    int pixelIdx = (y * width + x) * channels;
    for (int c = 0; c < 3; c++) {
        output[pixelIdx + c] = avgColor[maxIntensity][c] / maxCount;
    }
}

// CPU Function for Oil Painting Effect
void applyOilPaintingEffectCPU(Mat &image) {
    Mat result = image.clone();

    for (int y = 0; y < image.rows; y++) {
        for (int x = 0; x < image.cols; x++) {
            int hist[INTENSITY_LEVELS] = {0};
            int avgColor[INTENSITY_LEVELS][3] = {0};

            for (int dy = -RADIUS; dy <= RADIUS; dy++) {
                for (int dx = -RADIUS; dx <= RADIUS; dx++) {
                    int nx = min(max(x + dx, 0), image.cols - 1);
                    int ny = min(max(y + dy, 0), image.rows - 1);

                    Vec3b pixel = image.at<Vec3b>(ny, nx);
                    int intensity = (pixel[0] + pixel[1] + pixel[2]) / 3;
                    hist[intensity]++;
                    for (int c = 0; c < 3; c++) {
                        avgColor[intensity][c] += pixel[c];
                    }
                }
            }

            int maxIntensity = 0, maxCount = 0;
            for (int i = 0; i < INTENSITY_LEVELS; i++) {
                if (hist[i] > maxCount) {
                    maxCount = hist[i];
                    maxIntensity = i;
                }
            }

            for (int c = 0; c < 3; c++) {
                result.at<Vec3b>(y, x)[c] = avgColor[maxIntensity][c] / maxCount;
            }
        }
    }

    image = result;
}

// GPU Function for Oil Painting Effect
void applyOilPaintingEffectGPU(Mat &image) {
    int imgSize = image.rows * image.cols * 3;
    unsigned char *d_input, *d_output;
    hipMalloc(&d_input, imgSize);
    hipMalloc(&d_output, imgSize);
    hipMemcpy(d_input, image.data, imgSize, hipMemcpyHostToDevice);

    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((image.cols + BLOCK_SIZE - 1) / BLOCK_SIZE, (image.rows + BLOCK_SIZE - 1) / BLOCK_SIZE);

    hipEvent_t start, stop;
    float milliseconds = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    oilPaintingKernel<<<grid, block>>>(d_input, d_output, image.cols, image.rows, 3);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    cout << "CUDA Processing Time: " << milliseconds << " ms" << endl;

    hipMemcpy(image.data, d_output, imgSize, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}

int main() {
    Mat image = imread("input2.jpg");
    if (image.empty()) {
        cerr << "Error loading image!" << endl;
        return -1;
    }

    // CPU Timing
    Mat imageCPU = image.clone();
    auto startCPU = high_resolution_clock::now();
    applyOilPaintingEffectCPU(imageCPU);
    auto stopCPU = high_resolution_clock::now();
    auto durationCPU = duration_cast<milliseconds>(stopCPU - startCPU);
    cout << "CPU Processing Time: " << durationCPU.count() << " ms" << endl;

    // GPU Timing
    Mat imageGPU = image.clone();
    applyOilPaintingEffectGPU(imageGPU);

    // Save Results
    imwrite("img/output_oilpainting_cpu.jpg", imageCPU);
    imwrite("img/output_oilpainting_gpu.jpg", imageGPU);

    return 0;
}
